#include "hip/hip_runtime.h"
#include <QVector>
#include <QDebug>

#include <hip/hip_runtime.h>
#include "stdio.h"

#include "Grid.h"
#include "GridGpu.h"




__constant__ float Kernel[256];


__global__ void griddingKernel(TrajGpu devTraj, complexGpu *devKData, complexGpu *devGData, int gridSize)
{
    int blockWidth = ceilf((float)gridSize / gridDim.x);
    int blockHeight = ceilf((float)gridSize / gridDim.y);

    int blockStartX = blockWidth * blockIdx.x;
    int blockEndX = blockStartX + blockWidth;
    if (blockEndX > gridSize) blockEndX = gridSize;

    int blockStartY = blockHeight * blockIdx.y;
    int blockEndY = blockStartY + blockHeight;
    if (blockEndY > gridSize) blockEndY = gridSize;

    extern __shared__ complexGpu local_block[];

    int blockSize = blockWidth * blockHeight;
    for (int i = threadIdx.x; i < blockSize; i += blockDim.x) {
        local_block[i].real = 0;
        local_block[i].imag = 0;
    }
    __syncthreads();

    float kHW = 2;
    int klength = 256;

    int blockID = blockIdx.y * gridDim.x + blockIdx.x;
    kTraj *pTraj = (kTraj *)((char *)devTraj.trajData + devTraj.pitchTraj * blockID);

    for (int i = threadIdx.x; i < devTraj.trajWidth; i += blockDim.x) {
        kTraj traj = pTraj[i];
        if (traj.dcf == 0) break;

        float xCenter = (0.5f + traj.kx) * gridSize; // kx in (-0.5, 0.5)
        int xStart = ceilf(xCenter - kHW);
        int xEnd = floorf(xCenter + kHW);

        float yCenter = (0.5f + traj.ky) * gridSize; // ky in (-0.5, 0.5)
        int yStart = ceilf(yCenter - kHW);
        int yEnd = floorf(yCenter + kHW);

        if (xStart < blockStartX) xStart = blockStartX;
        if (xEnd > blockEndX - 1) xEnd = blockEndX - 1;

        if (yStart < blockStartY) yStart = blockStartY;
        if (yEnd > blockEndY - 1) yEnd = blockEndY - 1;

        int n = (yStart - blockStartY) * blockWidth + xStart - blockStartX;

        int dn = blockWidth - (xEnd - xStart) - 1;

        complexGpu data = devKData[traj.idx];

        float dataReal = traj.dcf * data.real;
        float dataImag = traj.dcf * data.imag;

        for (int y = yStart; y <= yEnd; y++) {
            float dy = y - yCenter;
            float dy2 = dy * dy;

            for (int x = xStart; x <= xEnd; x++) {
                float dx = x - xCenter;
                float dk = sqrtf(dy2 + dx * dx);

                if (dk < kHW) {
                    int ki = rintf(dk / kHW * (klength - 1));
                    // local_block[n].real += Kernel[ki] * dataReal;
                    // local_block[n].imag += Kernel[ki] * dataImag;
                    atomicAdd(&local_block[n].real, Kernel[ki] * dataReal);
                    atomicAdd(&local_block[n].imag, Kernel[ki] * dataImag);
                }
                n++;
            }
            n += dn;
        }
    }

    __syncthreads();

    for (int i = threadIdx.x; i < blockSize; i += blockDim.x) {
        int x = i % blockWidth + blockStartX;
        int y = i / blockWidth + blockStartY;
        if (x < blockEndX && y < blockEndY) {
            int idx = y  * gridSize + x;
            devGData[idx].real = local_block[i].real;
            devGData[idx].imag = local_block[i].imag;
        }
    }
}


hipError_t GridGpu::copyKernelData()
{
    QVector<float> kernelData = m_kernel.getKernelData();
    Q_ASSERT(kernelData.size() == 256);
    // Copy gridding kernel data
    hipMemcpyToSymbol(HIP_SYMBOL(Kernel), kernelData.data(), kernelData.size() * sizeof(float));

    return hipGetLastError();
}


hipError_t GridGpu::kernelCall(complexVector &kData)
{
    hipMemcpy(m_d_kData, kData.data(), kData.size() * sizeof(complexGpu), hipMemcpyHostToDevice);

    dim3 GridSize(m_gpuGridSize, m_gpuGridSize);
    griddingKernel<<<GridSize, m_threadsPerBlock, m_sharedSize>>>(m_d_Traj, m_d_kData, m_d_gData, m_gridSize);

    //std::complex<float> *p = gData.data();
    //qWarning() << p[0].real() << p[0].imag();

    return hipGetLastError();
}
