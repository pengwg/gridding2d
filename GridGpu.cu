#include <QVector>
#include <hip/hip_runtime.h>

#include "Grid.h"

__constant__ float kernelGpu[256];

hipError_t copyDataGpu(const QVector<float> &kernelData, const QVector< QVector<kData> > &dataPartition)
{
    Q_ASSERT(kernelData.size() == 256);
    hipMemcpyToSymbol(HIP_SYMBOL(kernelGpu), kernelData.data(), kernelData.size() * sizeof(float), hipMemcpyHostToDevice);

    int maxP = 0;

    for (int i = 0; i < dataPartition.size(); i++) {
        if (dataPartition[i].size() > maxP) maxP = dataPartition.size();
        // qWarning() << "Partition size:" << dataP.size();
    }


    return hipGetLastError();
}

hipError_t griddingGpu()
{
    return hipGetLastError();
}
