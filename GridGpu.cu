#include "hip/hip_runtime.h"
#include <QVector>
#include <QDebug>

#include <hip/hip_runtime.h>

#include "Grid.h"
#include "GridGpu.h"


int threadsPerBlock;
int gpuGridSize;


TrajGpu devTraj;
float *devKData;
float *devGData;


__constant__ float Kernel[256];


__global__ void griddingKernel(float *devKDataSet, float *devGDataSet, int gridSize)
{
    devGDataSet[0] = 1;
}

hipError_t copyKernel(const QVector<float> &kernelData)
{
    Q_ASSERT(kernelData.size() == 256);
    // Copy gridding kernel data
    hipMemcpyToSymbol(HIP_SYMBOL(Kernel), kernelData.data(), kernelData.size() * sizeof(float));

    return hipGetLastError();
}


hipError_t copyTraj(const QVector< QVector<kTraj> > &trajPartition)
{
    // Copy gridding k-trajectory data
    int maxP = 0;
    for (int i = 0; i < trajPartition.size(); i++) {
        if (trajPartition[i].size() > maxP) maxP = trajPartition[i].size();
        // qWarning() << "Partition" << i << trajPartition[i].size();
    }
    devTraj.trajWidth = maxP;

    hipMallocPitch(&devTraj.trajData, &devTraj.pitchTraj, maxP * sizeof(kTraj), trajPartition.size());
    qWarning() << "Partition pitch:" << devTraj.pitchTraj;

    for (int i = 0; i < trajPartition.size(); i++) {
        char *row = (char *)devTraj.trajData + i * devTraj.pitchTraj;
        hipMemcpy(row, trajPartition[i].data(), trajPartition[i].size() * sizeof(kTraj), hipMemcpyHostToDevice);
    }

    return hipGetLastError();
}

hipError_t mallocGpu(int kSize, int gSize)
{
    // Malloc k-space and gridding matrix data
    hipMalloc(&devKData, kSize * sizeof(float));
    hipMalloc(&devGData, gSize * sizeof(float));

    return hipGetLastError();
}

hipError_t griddingGpu(complexVector &kData, complexVector &gData, int gridSize)
{
    qWarning() << "In gridding GPU";
    hipMemcpy(devKData, kData.data(), kData.size() * sizeof(float), hipMemcpyHostToDevice);

    dim3 GridSize(gpuGridSize, gpuGridSize);
    griddingKernel<<<GridSize, threadsPerBlock>>>(devKData, devGData, gridSize);

    hipMemcpy(gData.data(), devGData, gData.size() * sizeof(float), hipMemcpyDeviceToHost);
    return hipGetLastError();
}
