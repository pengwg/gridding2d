#include "hip/hip_runtime.h"
#include <QVector>
#include <QDebug>

#include <hip/hip_runtime.h>
#include "stdio.h"

#include "Grid.h"
#include "GridGpu.h"


int threadsPerBlock;
int gpuGridSize;


TrajGpu devTraj;
complexGpu *devKData;
complexGpu *devGData;
int sharedSize;

__constant__ float Kernel[256];


__global__ void griddingKernel(TrajGpu devTraj, complexGpu *devKData, complexGpu *devGData, int gridSize)
{
    int blockWidth = ceilf((float)gridSize / gridDim.x);
    int blockHeight = ceilf((float)gridSize / gridDim.y);

    int blockStartX = blockWidth * blockIdx.x;
    int blockEndX = blockStartX + blockWidth;
    if (blockEndX > gridSize) blockEndX = gridSize;

    int blockStartY = blockHeight * blockIdx.y;
    int blockEndY = blockStartY + blockHeight;
    if (blockEndY > gridSize) blockEndY = gridSize;

    extern __shared__ complexGpu local_block[];

    int blockSize = blockWidth * blockHeight;
    for (int i = threadIdx.x; i < blockSize; i += blockDim.x) {
        local_block[i].real = 0;
        local_block[i].imag = 0;
    }
    __syncthreads();

    float kHW = 2;
    int klength = 256;

    int blockID = blockIdx.y * gridDim.x + blockIdx.x;
    kTraj *pTraj = (kTraj *)((char *)devTraj.trajData + devTraj.pitchTraj * blockID);

    for (int i = threadIdx.x; i < devTraj.trajWidth; i += blockDim.x) {
        kTraj traj = pTraj[i];
        if (traj.dcf == 0) break;

        float xCenter = (0.5f + traj.kx) * gridSize; // kx in (-0.5, 0.5)
        int xStart = ceilf(xCenter - kHW);
        int xEnd = floorf(xCenter + kHW);

        float yCenter = (0.5f + traj.ky) * gridSize; // ky in (-0.5, 0.5)
        int yStart = ceilf(yCenter - kHW);
        int yEnd = floorf(yCenter + kHW);

        if (xStart < blockStartX) xStart = blockStartX;
        if (xEnd > blockEndX - 1) xEnd = blockEndX - 1;

        if (yStart < blockStartY) yStart = blockStartY;
        if (yEnd > blockEndY - 1) yEnd = blockEndY - 1;

        int n = (yStart - blockStartY) * blockWidth + xStart - blockStartX;

        int dn = blockWidth - (xEnd - xStart) - 1;

        complexGpu data = devKData[traj.idx];

        float dataReal = traj.dcf * data.real;
        float dataImag = traj.dcf * data.imag;

        for (int y = yStart; y <= yEnd; y++) {
            float dy = y - yCenter;
            float dy2 = dy * dy;

            for (int x = xStart; x <= xEnd; x++) {
                float dx = x - xCenter;
                float dk = sqrtf(dy2 + dx * dx);

                if (dk < kHW) {
                    int ki = rintf(dk / kHW * (klength - 1));
                    //local_block[n].real += Kernel[ki] * dataReal;
                    //local_block[n].imag += Kernel[ki] * dataImag;
                    atomicAdd(&local_block[n].real, Kernel[ki] * dataReal);
                    atomicAdd(&local_block[n].imag, Kernel[ki] * dataImag);
                }
                n++;
            }
            n += dn;
        }
    }

    __syncthreads();

    for (int i = threadIdx.x; i < blockSize; i += blockDim.x) {
        int x = i % blockWidth + blockStartX;
        int y = i / blockWidth + blockStartY;
        if (x < blockEndX && y < blockEndY) {
            int idx = y  * gridSize + x;
            devGData[idx].real = local_block[i].real;
            devGData[idx].imag = local_block[i].imag;
        }
    }
}

hipError_t copyKernel(const QVector<float> &kernelData)
{
    Q_ASSERT(kernelData.size() == 256);
    // Copy gridding kernel data
    hipMemcpyToSymbol(HIP_SYMBOL(Kernel), kernelData.data(), kernelData.size() * sizeof(float));

    return hipGetLastError();
}


hipError_t copyTraj(const QVector< QVector<kTraj> > &trajPartition)
{
    // Copy gridding k-trajectory data
    int maxP = 0;
    for (int i = 0; i < trajPartition.size(); i++) {
        if (trajPartition[i].size() > maxP) maxP = trajPartition[i].size();
        // qWarning() << "Partition" << i << trajPartition[i].size();
    }
    devTraj.trajWidth = maxP;

    hipMallocPitch(&devTraj.trajData, &devTraj.pitchTraj, maxP * sizeof(kTraj), trajPartition.size());
    hipMemset(devTraj.trajData, 0, devTraj.pitchTraj * trajPartition.size());
    qWarning() << "Partition pitch:" << devTraj.pitchTraj;

    for (int i = 0; i < trajPartition.size(); i++) {
        char *row = (char *)devTraj.trajData + i * devTraj.pitchTraj;
        hipMemcpy(row, trajPartition[i].data(), trajPartition[i].size() * sizeof(kTraj), hipMemcpyHostToDevice);
    }

    return hipGetLastError();
}

hipError_t mallocGpu(int kSize, int gSize)
{
    // Malloc k-space and gridding matrix data
    hipMalloc(&devKData, kSize * sizeof(complexGpu));
    hipMalloc(&devGData, gSize * sizeof(complexGpu));

    return hipGetLastError();
}

hipError_t griddingGpu(complexVector &kData, complexVector &gData, int gridSize)
{
    hipMemcpy(devKData, kData.data(), kData.size() * sizeof(complexGpu), hipMemcpyHostToDevice);

    dim3 GridSize(gpuGridSize, gpuGridSize);
    griddingKernel<<<GridSize, threadsPerBlock, sharedSize>>>(devTraj, devKData, devGData, gridSize);

    //hipMemcpy(gData.data(), devGData, gData.size() * sizeof(complexGpu), hipMemcpyDeviceToHost);
    //std::complex<float> *p = gData.data();
    //qWarning() << p[0].real() << p[0].imag();

    return hipGetLastError();
}
